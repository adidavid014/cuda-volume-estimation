
#include <hip/hip_runtime.h>
#include <iostream>
#include <vector>
#include <fstream>
#include <hiprand/hiprand_kernel.h>
#include <cmath>

using namespace std;

__global__ void computeHistogram(int d, int samples, int bins, double *histogram, int *count){
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    if(idx < samples){
        hiprandState state;
        hiprand_init(1234, idx, 0, &state);

        double sum = 0;
        for(int j = 0; j < d; ++j){
            double value = hiprand_uniform_double(&state) * 2 - 1;
            sum += value * value;
        }
        if(sum <= 1){
            atomicAdd(count, 1);
            double distance = 1-sqrt(sum);
            int bin = min(static_cast<int>(distance * bins), bins-1);
            atomicAdd(&histogram[bin], 1.0);
        }
    }
}

int main() {
    const int D = 16;
    const int samples = 2000000;
    const int bins = 100;
    vector<double> histogramHost(bins, 0);
    double *histogramDevice;
    hipMalloc(&histogramDevice, bins * sizeof(double));
    int countHost = 0;
    int *countDevice;
    hipMalloc(&countDevice, sizeof(int));
    const int blockSize = 256;
    const int numBlocks = (samples+blockSize-1)/blockSize;

    //adding matplot output
    ofstream outFile("histogram_data.txt");

    for(int d = 2; d <= D; ++d){
        hipMemset(histogramDevice, 0, bins * sizeof(double));
        hipMemset(countDevice, 0, sizeof(int));
        computeHistogram<<<numBlocks, blockSize>>>(d, samples, bins, histogramDevice, countDevice);
        hipDeviceSynchronize();

        //finding driver error 
        hipError_t error = hipGetLastError();
        if (error != hipSuccess) {
            cout << "CUDA error: " << hipGetErrorString(error) << endl;
            break;
        }
        hipMemcpy(&countHost, countDevice, sizeof(int), hipMemcpyDeviceToHost);
        hipMemcpy(histogramHost.data(), histogramDevice, bins * sizeof(double), hipMemcpyDeviceToHost);

        if(countHost > 0){
            outFile << d;
            for (auto h : histogramHost) {
                outFile << " " << h / countHost;
            }
            outFile << endl;
        }
    }
    outFile.close();
    hipFree(histogramDevice);
    hipFree(countDevice);
    return 0;
}